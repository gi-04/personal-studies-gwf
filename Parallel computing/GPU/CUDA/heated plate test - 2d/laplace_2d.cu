
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

void print_2d_array(int Nx,int Ny,int k,double *M);
__global__
void initialize(int size,double *M);
__device__
void solver(int Nx,int Ny,double *sol,int k,double beta);
__global__
void physical_boundary_conditions(int Nxs,int Nys,double *sol,double T_left,
																	double T_right,double T_up,double T_down);
__global__
void phantom_boundary_conditions(int Nxs,int Nys,double *sol);
__device__
void save_sol_old(int Nxs,int Nys,double *sol,double *sol_old);
__device__
void get_residuals(int Nxs,int Nys,double *sol,double *sol_old,double *res);
double sum_residuals(int Nxs,int Nys,double *res);
__global__ 
void iterate_once(int Nxs,int Nys,double *sol,double *sol_old,double *res,
									double beta);
void parallel_solver(int Nxs,int Nys,double *sol,double *sol_old,double *res,
										 int iter,double beta,double eps,int blocksPerGrid,
										 int threadsPerBlock);
										 
int main(){
	time_t start,end;
	time(&start);
	
	// Problem parameters
  double Lx = 1., Ly = 1.;
  double T_down = 0., T_up = 0.;
  double T_left = 50., T_right = 50.;

  // Mesh parameters
  int Nx = 100, Ny = 100; // Complete domain (even values)
  double deltaX = Lx/((double) Nx);
  double deltaY = Ly/((double) Ny);
  double beta = deltaX/deltaY;
	
	// Numeric solution configuration
  int iter = 50000;
  double eps = 1e-3;
	
	// Allocate relevant arrays in GPU memory
	Nx = (int) (Nx/2)*2; // Ensure values are even
	Ny = (int) (Ny/2)*2;
	int Nxs = Nx/2+1; // Dimensions for the submatrices
	int Nys = Ny/2+1;
	double *sol = new double[4*Nxs*Nys];
	double *sol_old = new double[4*(Nxs-2)*(Nys-2)];
	double *res = new double[4*(Nxs-2)*(Nys-2)];
	hipMallocManaged(&sol,sizeof(double)*4*Nxs*Nys);
	hipMallocManaged(&sol_old,sizeof(double)*4*(Nxs-2)*(Nys-2));
	hipMallocManaged(&res,sizeof(double)*4*(Nxs-2)*(Nys-2));
	
	// Initialize arrays
	int threadsPerBlock = 256;
	int blocksPerGrid = (4*Nxs*Nys+threadsPerBlock-1)/threadsPerBlock;
	// dim3 threadsPerBlock(threadsPerBlock_,threadsPerBlock_,1);
	// dim3 blocksPerGrid(blocksPerGrid_,blocksPerGrid_,1);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*Nxs*Nys,sol);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*(Nxs-2)*(Nys-2),sol_old);
	initialize<<<blocksPerGrid,threadsPerBlock>>>(4*(Nxs-2)*(Nys-2),res);
	hipDeviceSynchronize();
	
  // Insert physical boundary conditions
	physical_boundary_conditions<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol,
															T_left,T_right,T_up,T_down);
	hipDeviceSynchronize();
	
  // Obtain solution
  parallel_solver(Nxs,Nys,sol,sol_old,res,iter,beta,eps,blocksPerGrid,
									threadsPerBlock);
	hipDeviceSynchronize();

  // Print results to file
  FILE *results;
  results = fopen("results.txt","w");
  fprintf(results,"%.8f %.8f ",Lx,Ly);
  for(int i=2;i<Nx;i++)
    fprintf(results,"%.8f ",0.);
  fprintf(results,"\n");
	
	for(int j=Nys-1;j>0;j--){
		for(int i=0;i<Nxs-1;i++)
			fprintf(results,"%.8f ",sol[0*Nxs*Nys+Nys*j+i]);
		for(int i=1;i<Nxs;i++)
			fprintf(results,"%.8f ",sol[1*Nxs*Nys+Nys*j+i]);
		
		fprintf(results,"\n");
	}
	for(int j=Nys-2;j>=0;j--){
		for(int i=0;i<Nxs-1;i++)
			fprintf(results,"%.8f ",sol[2*Nxs*Nys+Nys*j+i]);
		for(int i=1;i<Nxs;i++)
			fprintf(results,"%.8f ",sol[3*Nxs*Nys+Nys*j+i]);
		
		fprintf(results,"\n");
	}
  fclose(results);
	
	// Free memories
	hipFree(sol);
	hipFree(sol_old);
	hipFree(res);
	// free(sol_host);
	// free(sol_old_host);
	// delete[] sol;
	// delete[] sol_old;
	
	time(&end);
	double time_taken = double(end - start);
	printf("\n%f s\n",time_taken);
	
	puts("end");

  return 0;
}

void print_2d_array(int Nx,int Ny,int k,double *M){
  for(int j=Ny-1;j>=0;j--){
    for(int i=0;i<Nx;i++)
      printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    putchar('\n');
  }
}

// __device__
// void dev_print_2d_array(int Nx,int Ny,int k,double *M){
  // for(int j=Ny-1;j>=0;j--){
    // for(int i=0;i<Nx;i++)
      // printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    // printf("\n");
  // }
// }

// void print_2d_array(int Nx,int Ny,int k,double *M){
  // for(int j=0;j<Ny;j++){
    // for(int i=0;i<Nx;i++)
      // printf("%f ",M[k*Nx*Ny+Ny*j+i]);

    // putchar('\n');
  // }
// }

__global__
void initialize(int size,double *M){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
  for(int i=index;i<size;i+=stride){
    M[i] = 0;
	}
}

__device__
void solver(int Nx,int Ny,double *sol,int k,double beta){
	if(threadIdx.x == k){
		for(int j=1;j<Ny-1;j++){
			for(int i=1;i<Nx-1;i++){
				sol[k*Nx*Ny+Ny*j+i] = 1./(2.*(1.+beta*beta))*(sol[k*Nx*Ny+Ny*j+(i+1)] + 
															sol[k*Nx*Ny+Ny*j+(i-1)] + 
															beta*beta*sol[k*Nx*Ny+Ny*(j+1)+i] + 
															beta*beta*sol[k*Nx*Ny+Ny*(j-1)+i]);
			}
		}
	}
}

// Regions are considered as below:
// | [0] [1] |
// | [2] [3] | 
__global__
void physical_boundary_conditions(int Nxs,int Nys,double *sol,double T_left,
																	double T_right,double T_up,double T_down){
	
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	for(int j=index;j<Nys-1;j+=stride){
		sol[0*Nxs*Nys+Nys*j+0] = T_left;
		sol[2*Nxs*Nys+Nys*j+0] = T_left;
		sol[1*Nxs*Nys+Nys*j+(Nxs-1)] = T_right;
		sol[3*Nxs*Nys+Nys*j+(Nxs-1)] = T_right;
  }
  for(int i=index;i<Nxs-1;i+=stride){
		sol[2*Nxs*Nys+Nys*0+i] = T_down;
		sol[3*Nxs*Nys+Nys*0+i] = T_down;
		sol[0*Nxs*Nys+Nys*(Nys-1)+i] = T_up;
		sol[1*Nxs*Nys+Nys*(Nys-1)+i] = T_up;
  }
}

__global__
void phantom_boundary_conditions(int Nxs,int Nys,double *sol){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	// Vertical
	for(int j=index;j>0&&j<Nys;j+=stride){
		// Left to right
		sol[3*Nxs*Nys+Nys*j+0] = sol[2*Nxs*Nys+Nys*j+(Nxs-2)];
		sol[1*Nxs*Nys+Nys*(j-1)+0] = sol[0*Nxs*Nys+Nys*(j-1)+(Nxs-2)];
		
		// Right to left
		sol[2*Nxs*Nys+Nys*j+(Nxs-1)] = sol[3*Nxs*Nys+Nys*j+1];
		sol[0*Nxs*Nys+Nys*(j-1)+(Nxs-1)] = sol[1*Nxs*Nys+Nys*(j-1)+1];
	}
	
	// Horizontal
	for(int i=index;i>0&&i<Nxs;i+=stride){
		// Lower to upper
		sol[0*Nxs*Nys+Nys*0+i] = sol[2*Nxs*Nys+Nys*(Nys-2)+i];
		sol[1*Nxs*Nys+Nys*0+(i-1)] = sol[3*Nxs*Nys+Nys*(Nys-2)+(i-1)];
		
		// Upper to lower
		sol[2*Nxs*Nys+Nys*(Nys-1)+i] = sol[0*Nxs*Nys+Nys*1+i];
		sol[3*Nxs*Nys+Nys*(Nys-1)+(i-1)] = sol[1*Nxs*Nys+Nys*1+(i-1)];	
	}
}

__device__
void save_sol_old(int Nxs,int Nys,double *sol,double *sol_old){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	int k;
	int row;
	int col;
	
	for(int i=index;i<(Nxs-2)*(Nys-2)*4;i+=stride){
		k = i/((Nxs-2)*(Nys-2));
		row = (i - k*(Nxs-2)*(Nys-2))/(Nxs-2);
		col = i - k*(Nxs-2)*(Nys-2) - (Nys-2)*row;
		sol_old[i] = sol[k*Nxs*Nys+Nys*(row+1)+(col+1)];
	}
}

__device__
void get_residuals(int Nxs,int Nys,double *sol,double *sol_old,double *res){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	int k;
	int row;
	int col;
	for(int i=index;i<(Nxs-2)*(Nys-2);i+=stride){
		k = i/((Nxs-2)*(Nys-2));
		row = (i - k*(Nxs-2)*(Nys-2))/(Nxs-2);
		col = i - k*(Nxs-2)*(Nys-2) - (Nys-2)*row;
		res[i] = fabs(sol[k*Nxs*Nys+Nys*(row+1)+(col+1)] - 
							 sol_old[i]);
	}
}

double sum_residuals(int Nxs,int Nys,double *res){
	double res_sum = 0.;
	for(int i=0;i<Nxs*Nys*4;i++)
		res_sum += res[i];
	
	return res_sum;
}

__global__ 
void iterate_once(int Nxs,int Nys,double *sol,double *sol_old,double *res,
									double beta){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x;
	save_sol_old(Nxs,Nys,sol,sol_old); // Save current values for convergence checking
	for(int k=index;k<4;k+=stride){
		solver(Nxs,Nys,sol,k,beta);
	}
	get_residuals(Nxs,Nys,sol,sol_old,res);
}

void parallel_solver(int Nxs,int Nys,double *sol,double *sol_old,double *res,
										 int iter,double beta,double eps,int blocksPerGrid,
										 int threadsPerBlock){
	double res_val;
	for(int loop=0;loop<iter;loop++){
		// Calculate one iteration for each region
		iterate_once<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol,sol_old,res,
								beta);
		hipDeviceSynchronize();
		
		// for(int k=0;k<4;k++){
			// printf("%d\n",k);
			// print_2d_array(Nxs,Nys,k,sol);
			// putchar('\n');
			// print_2d_array(Nxs-2,Nys-2,k,sol_old);
			// putchar('\n');
			// print_2d_array(Nxs-2,Nys-2,k,res);
			// putchar('\n');
		// }
		// getchar();
		
		// Check convergence
		res_val = sum_residuals(Nxs-2,Nys-2,res);
		if(res_val <= eps){
			puts("Convergence!");
			break;
		}
		
		printf("Iteration %d | Residuals = %f\n",loop+1,res_val);
		
		// Update phantom boundary conditions
		phantom_boundary_conditions<<<blocksPerGrid,threadsPerBlock>>>(Nxs,Nys,sol);
		hipDeviceSynchronize();
	}
}